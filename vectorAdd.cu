
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <time.h>


using namespace std;




void init(int *a,int n){
	
	for(int i=0;i<n;i++){
		a[i]=i;
	}

}

bool verify(int *a,int *b,int *c,int n){

	int s=0;
	for(int i=0;i<n;i++){
		s=a[i]+b[i];
		if(s!=c[i])
		return false;
	}
	return true;
	
}

__global__ void vectorAdd(int *a,int *b,int *c,int n){

	int i=blockIdx.x * blockDim.x + threadIdx.x;
	int temp=0;
	if(i<n){
		temp+=a[i]+b[i];
	}
	c[i]=temp;

}

int main(){

	int N=1<<10;
	int *a,*b,*c;
	
	hipMallocManaged(&a,N*sizeof(int));
	hipMallocManaged(&b,N*sizeof(int));
	hipMallocManaged(&c,N*sizeof(int));
	init(a,N);
	init(b,N);
	
	int thread=16;
	int blockx=(N+thread-1)/thread;
	
	clock_t start,end;
	start=clock();
	vectorAdd<<<blockx,thread>>>(a,b,c,N);
	end=clock();
	
	hipDeviceSynchronize();
	double tp=end-start;
	
	start=clock();
	int res=verify(a,b,c,N);
	end=clock();
	
	double ts=end-start;
	
	if(verify(a,b,c,N)){
		cout<<"Verified "<<endl;
		cout<<"Speedup "<<ts/tp<<endl;
	}
	else{
		cout<<"UnSucessful";
	}
	
	return 0;


}
