
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <time.h>

using namespace std;


__global__ void matrixMul(int *a,int *b,int *c,int N){
    // Calculate the global row and column for each thread

    int row=blockIdx.y * blockDim.y + threadIdx.y;
    int col=blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if(row<N && col<N){
        // Accumulate a partial result
        int temp=0;
        for(int i=0;i<N;i++){
            temp+=a[row*N + i] * b[i*N + col];
        }

        //Write back the result
        c[row*N + col]= temp;



    }

    

}

void init_matrix(int *m, int N){
    for(int i=0;i< N*N;i++){
        m[i]=rand()%100;
    }
}

void verify_result(int *a, int *b,int *c, int N){
    int tmp;

    for(int i=0;i<N;i++){
        
        for(int j=0;j<N;j++){
            tmp=0;
            for(int k=0;k<N;k++){
                tmp+=a[i*N + k] * b[k*N + j];
            }

            //check each result
            assert(tmp==c[i*N+j]);


        }
                             
    }

}

int main(){
     
    int N = 1<<10;
    size_t bytes = N*N* sizeof(int);

    

    int *a,*b,*c;
    hipMallocManaged(&a,bytes);
    hipMallocManaged(&b,bytes);
    hipMallocManaged(&c,bytes);

    // Initialize our matrics
    init_matrix(a,N);
    init_matrix(b,N);

   

    int  threads=16;
    int blocks= (N + threads - 1) / threads;

   
    dim3 THREADS(threads,threads);
    dim3 BLOCKS(blocks,blocks);

    clock_t start, end;


    start = clock();
    matrixMul<<<BLOCKS, THREADS>>>(a,b,c,N);
    hipDeviceSynchronize();
    end = clock();

    float GPUTime = ((float)(end-start))/CLOCKS_PER_SEC;
    cout<<"Parallel time "<<GPUTime<<endl;
    
    start = clock();
    //Verify the result
    verify_result(a,b,c,N);
    end = clock();

    float CPUTime=((float)(end-start))/CLOCKS_PER_SEC;

    cout<<"Sequential time "<<CPUTime<<endl;
    cout<<"Speed up "<<(CPUTime/GPUTime)<<endl;
    cout<<"Program Completed Successfully";
}
